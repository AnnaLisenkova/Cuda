#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <map>
#include <fstream>

using namespace std;

#define delta                10
#define rows                     500
#define columns                  500


int* findBarrier(int x, int y, int * Map[columns]){
        //y-координаты препятствий
        int *yCoordinates = new int [columns];
        //текущее значение разности м-у двумя точками по вертикали
        int currDelta = 0;

        for(int i = 0; i < columns; i++){
                //рассматриваем область выше параллели, на которой стоит робот
                for(int j = y; j > 0; j--){
                        currDelta = Map[j][i] - Map[j-1][i];
                        //если текущая разность больше дельты, то запоминаем у-координату
                        if( ( currDelta >= 0 ? currDelta : currDelta*-1 ) > delta){
                                yCoordinates[i] = j-1;
                                break;
                        }
                }
        }
        return yCoordinates;
}

__global__ void SomeKernel(int* res, int* data, int col, int row,int y, int step)
{
   unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
        //Считаем идентификатор текущего потока
   int currDelta = 0;
   for (int i=step*threadId; (i<(threadId+1)*step) && (i < col); i++) //Работа со столбцами по потокам
   {
           for (int j = y; j > 0; j--) //Здесь работа со строками
           {
                        currDelta = data[i + j*row] - data[i + (j-1)*row];
                        //если текущая разность больше дельты, то запоминаем у-координату
                        if( ( currDelta >= 0 ? currDelta : currDelta*-1 ) > 10){
                                res[i] = j-1;
                                break;
                        }
           }
   }
}

//int argc, char* argv[]
int main(int argc, char* argv[]){
        map<int,float> Results;

        int numbOfBlock = 1;
        int numbOfThread = 1;
        for (int i = 0; i < columns; i++ )
        {
			if (i&1)
				numbOfThread++;
			else
				numbOfBlock++;
                //for(int numbOfThread = 1; numbOfThread <= columns; numbOfThread++){
                //if(columns % numbOfBlock == 0){
                        //numbOfThread = 1;
        //if (argc > 1)
        //      numbOfBlock = atoi(argv[1]);
        //else
        //      numbOfBlock = 1;
        //if (argc > 2)
        //      numbOfThread = atoi(argv[2]);
        //else
        //      numbOfThread = 1;
        //левая и правая границы высот для генерации
        const int r_left = -20, r_right = 20;
        //Координаты робота на карте
        //int x = rows - 1;
        int y = columns - 1;

        //Карта высот
        int **Map = new int* [rows];
    int* resH = (int*)malloc(rows*columns * sizeof(int));
        for (int i=0; i<columns; i++)
                resH[i] = 0;

        //Заполнение карты случайыми высотами
        for(int i = 0; i < rows; i++){
                Map[i] = new int [columns];

                for(int j = 0; j < columns; j++){
                        //if(j!=0)
                                Map[i][j] = rand()%(r_left - r_right) + r_left;
                        //else
                                //Map[i][j] = 20;
                }
        }
        //Помещаем двумерный массив высот в одномерный
        int* dataH = (int*)malloc(columns * rows * sizeof(int));
   for (int i=0; i<columns; i++)
           for (int j=0; j<rows; j++)
                        dataH[columns*i + j] = Map[i][j];





hipEvent_t start, stopCopyTo, stopWork, stopCopyFrom;
hipEventCreate(&start);
hipEventCreate(&stopCopyTo);
hipEventCreate(&stopWork);
hipEventCreate(&stopCopyFrom);





   int* dataDevice;
   int* resDevice;
//Выделяем память на GPU под созданный массив
   hipMalloc((void**)&dataDevice, (rows * columns) * sizeof(int));
   hipMalloc((void**)&resDevice, (columns) * sizeof(int));
// Копирование исходных данных в GPU для обработки


hipEventRecord(start);
   hipMemcpy(dataDevice, dataH, (rows * columns) * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(resDevice, resH, (columns)*sizeof(int), hipMemcpyHostToDevice);

   dim3 threads = dim3(numbOfThread);
   dim3 blocks = dim3(numbOfBlock);

 hipEventRecord(stopCopyTo);

           SomeKernel<<<blocks, threads>>>( resDevice,
                                                                                dataDevice,
                                                                                columns,
                                                                                rows,
                                                                                y,
                                        (rows * columns)/(numbOfBlock*numbOfThread));


hipEventRecord(stopWork);

hipMemcpy(dataH, dataDevice, (rows * columns) * sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(resH, resDevice, (columns) * sizeof(int), hipMemcpyDeviceToHost);

hipEventRecord(stopCopyFrom);
cout << "Result vector:  ";
        for (int i=0; i<5; i++)
        {
                cout << resH[i] << " ";
        }

      cout<<'\t';


        for(int i = 0; i < columns; i++){
                delete[] Map[i];
        }

float t1,t2,t3;
hipEventElapsedTime(&t1, start, stopCopyTo);
hipEventElapsedTime(&t2, stopCopyTo, stopWork);
hipEventElapsedTime(&t3, stopWork, stopCopyFrom);

        //cout<<"Threads: "<< numbOfBlock*numbOfThread <<"\tTime: "<<t2<<endl;
        Results.insert(pair<int,float>(numbOfBlock*numbOfThread,t2));
   
}
map<int,float>::iterator it;
ofstream fout("tt1.txt");
  for (it = Results.begin(); it != Results.end(); ++it)///вывод на экран
  {
     fout << it->first << ' ' << it->second << endl;
  }
  fout.close();
//cout << "Количество точек: \t\t" << columns*rows << endl;
//cout << "Количество потоков: \t\t" << numbOfBlock*numbOfThread << endl;
//cout << "Время копирования на GPU: \t" << t1 << endl;
//cout << "Время выполенния: \t\t" << t2 << endl;
//cout << "Время копирования с GPU: \t"  << t3 << endl;

        return 0;
}

